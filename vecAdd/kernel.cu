﻿
#include "hip/hip_runtime.h"


#include <math.h>
#include <stdio.h>

hipError_t vecAdd(float* A_h, float* B_h, float* C_h, int n);

__global__ void vecAddKernel(float* A, float* B, float* C, int n)
{
    int i = threadIdx.x + blockDim.x * blockIdx.x;
    if (i < n)
    {
        C[i] = A[i] + B[i];
    }
}

int main()
{
    const int n = 10;

    float a[n] = {1.1, 2.2, 3.3, 4.4, 5.5, 6.6, 7.7, 8.8, 9.9, 10.1};
    float b[n] = { 1.1, 2.2, 3.3, 4.4, 5.5, 6.6, 7.7, 8.8, 9.9, 10.1};
    float c[n] = {};
    hipError_t cudaStatus = vecAdd(a, b, c, n);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addWithCuda failed!");
        return 1;
    }
    hipDeviceReset();
    
    int i = 0;
    do 
    {
        printf("%.1f, ", c[i]);
        ++i;
    } while (i < n - 1);
    
    printf("%.1f", c[n-1]);

    return 0;
}



hipError_t vecAdd(float* A_h, float* B_h, float* C_h, int n)
{
    int size = n * sizeof(float);
    float *A_d, *B_d, * C_d;

    hipError_t cudaStatus;

    cudaStatus = hipSetDevice(0);
    //Allocate device memory for A, B, C
    hipMalloc((void**)&A_d, size);
    hipMalloc((void**)&B_d, size);
    hipMalloc((void**)&C_d, size);

    //copy memory(host=A_h, B_h) to memory(device=A_d, B_d)
    hipMemcpy(A_d, A_h, size, hipMemcpyHostToDevice);
    hipMemcpy(B_d, B_h, size, hipMemcpyHostToDevice);

    //<<<dimGrid, dimBlock>>>
    vecAddKernel<<< ceil(n / 256.0), 256 >>>(A_d, B_d, C_d, n);
    
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
    }
    
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
    }

    //copy result(memory) from Device to Host
    hipMemcpy(C_h, C_d, size, hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
    }

    hipFree(A_d);
    hipFree(B_d);
    hipFree(C_d);

    return cudaStatus;
}

